#include "hip/hip_runtime.h"
//pass
//--blockDim=32 --gridDim=2

#include <hip/hip_runtime.h>

using namespace cooperative_groups;

__global__ void race (int* A)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  grid_group g = this_grid();
  thread_block t = this_thread_block();
  int idx = blockDim.x * bid + tid;

  int temp = A[idx + 1];
  synchronize(g);
  A[idx] = temp;
  
  synchronize(g);
  if (bid == 0)
  {
    temp = A[idx + 1];
    synchronize(t);
    A[idx] = temp;
  }
}
